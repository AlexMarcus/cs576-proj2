#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <algorithm>
#include <ctime>
#include <cmath>
#include <math.h>

#define NUM (1<<12)
#define U 2
#define V 16

using namespace std;

typedef struct{
        float x,y;
} point;

float ** generate_set(point a, point b, point c, point *points,point *all_points);
float get_distance(point a, point b);
__global__ void trilateration(point *a, point *b, point *c, float ** dv, point * pts);



int main(int argc, char *argv[]){
    srand(time(NULL));
    cout << NUM << endl;
    //point *results =(point *) malloc((NUM/4) * (sizeof(point)));
    point *points =(point *) malloc((NUM/4) * (sizeof(point)));
    point *all_points =(point *) malloc((NUM) * (sizeof(point)));

    point a = {3.4,-2.4};
    point b = {5.6,1.23};
    point c = {-3.8,5.4};
    
    float ** distance_vector = generate_set(a,b,c,points, all_points);

    float ** dv;
    point * da;
    point * db;
    point * dc;
    point * pts;

    /*
    hipMalloc(&da, sizeof(point *));
    hipMalloc(&db, sizeof(point *));
    hipMalloc(&dc, sizeof(point *));
    hipMalloc((void **)&pts, (NUM/4) * sizeof(point));
    hipMalloc((void **)&dv, NUM*sizeof(float *));

    for(int i = 0; i < NUM; i++){
    	    hipMalloc(&dv[i], 3*sizeof(float));
    }

    hipMemcpy(dv, distance_vector, NUM * sizeof(float*), hipMemcpyHostToDevice);
    hipMemcpy(da, &a, sizeof(point),hipMemcpyHostToDevice);
    hipMemcpy(db, &b, sizeof(point),hipMemcpyHostToDevice);
    hipMemcpy(dc, &c, sizeof(point),hipMemcpyHostToDevice);
    */

    hipMallocManaged(&da, sizeof(point *));
    hipMallocManaged(&db, sizeof(point *));
    hipMallocManaged(&dc, sizeof(point *));
    hipMallocManaged(&pts, (NUM) * sizeof(point));
    hipMallocManaged(&dv, NUM * sizeof(float *));
    for(int i = 0; i < NUM; i++){
	hipMallocManaged(&dv[i], 3*sizeof(float));
    }

    *da = a;
    *db = b;
    *dc = c;
    for(int i = 0; i < NUM; i++){
	for(int j = 0; j < 3; j++){
		dv[i][j] = distance_vector[i][j];
	}
    }

    point guard = {3.4, -2.4};
    point center = {1,1};
    cout << "HERE " << get_distance(guard, center) << endl;

    trilateration<<<U,V>>>(da,db,dc,dv,pts);
    hipDeviceSynchronize();
    
    //hipMemcpy(results, pts, (NUM/4) * sizeof(point),hipMemcpyDeviceToHost);

    /*for(int i = 0; i < NUM/4; i++){
	if(results[i].x != 0)
		cout << results[i].x << ", " << results[i].y << "\n";
    }*/

    for(int i = 0; i < 20; i++){
		cout << pts[i].x << ", " << pts[i].y << " | Actual point: " << all_points[i].x << ", " << all_points[i].y << "\n";
    }

	/*
	first points
	0.170442, -0.212715
	0.642852, -0.825177
	1.1408, -1.19354
	1.42159, -1.6076
	1.62658, -2.18829
	1.77826, -2.66518
	2.17155, -2.97643
	2.65642, -3.23502
	3.06065, -3.62799
	3.42531, -4.1356
	3.96662, -4.42709
	4.41263, -4.69845
	4.93671, -4.96385
	5.46347, -5.28732
	6.06384, -5.70538
	6.48004, -6.27095
	7.05871, -6.8165
	7.63242, -7.21735
	8.05061, -7.52002
	8.4322, -7.82146
	8.73583, -8.10203
	9.03263, -8.45821
	9.40358, -8.86348
	9.6679, -9.13161
 

	*/

    free(points);
    hipFree(da);
    hipFree(db);
    hipFree(dc);
    hipFree(points);
    for(int i = 0; i < NUM; i++)
            hipFree(dv[i]);
    hipFree(dv);

    return 0;    
}

float ** generate_set(point a, point b, point c, point *points, point *all_points){

    float ** dist = (float **) malloc(NUM * sizeof(float *)); 
    int i,j;
    for(j = 0; j < NUM; j++){
    	  dist[j] = (float *) malloc(3 * sizeof(float));
	  for(i = 0; i < 3; i++){
	  	dist[j][i] = 0;
	}	     
    }

    srand(time(NULL));
    float x_ave = 0, y_ave = 0;
    point next;
    next.x = 0;
    next.y = 0;
    for(i = 0; i < NUM; i++){
    	  dist[i][0] = get_distance(a,next);
	  dist[i][1] = get_distance(b,next);
	  dist[i][2] = get_distance(c,next);
	  all_points[i] = next;

	  //cout << dist[i][0] << "," << dist[i][1] << "," << dist[i][2] << endl;
	  
	  if(i%4 == 0 && i != 0){
	  	point t;
		t.x = x_ave/4;
		t.y = y_ave/4;
		points[(i/4)-1] = t;
		x_ave = 0;
		y_ave = 0;
		//if(i < 100) cout << t.x << ", " << t.y << endl;
	  }	  

	  x_ave	+= next.x;
	  y_ave	+= next.y;

	  //get new point
	  float temp = (rand() % 20000);
	  float delta_x = (temp / 100000);
	  temp = (rand() % 20000);
	  float delta_y = (temp / 100000) - .2;
	  
	  next.x += delta_x;
	  next.y += delta_y;
	  
    }

    return dist; 
    	  
}

float get_distance(point a, point b){
      float distance = sqrt((pow((a.x - b.x),2) + pow((a.y - b.y),2)));
      return distance;
}

float norm(point p){
	return pow(pow(p.x,2) + pow(p.y,2), .5);
}

__global__ void trilateration(point *a, point *b, point *c, float ** dv, point * pts){

	   int i = blockIdx.x * blockDim.x + threadIdx.x;
		   
	   int j;
	   for(j =0; j < ((NUM)/(U*V*4));j++){
	   	 float ave_y = 0, ave_x = 0;
	   	        float xa = a->x;
		 	float ya = a->y;
	   	 	float xb = b->x;
	   	 	float yb = b->y;
	   	 	float xc = c->x;
	   	 	float yc = c->y;
	   	 	float ra = dv[i+ j*(U*V)][0];
	   	 	float rb = dv[i+ j*(U*V)][1];
	   	 	float rc = dv[i+ j*(U*V)][2];

			float numerator = ((xb - xa) * (xc * xc + yc * yc - rc*rc) +
				(xa - xc) * (xb * xb + yb * yb - rb * rb) +
				(xc - xb) * (xa * xa + ya * ya - ra * ra));
	   		float denominator = (2 * (yc *(xb - xa) + yb * (xa - xc) + ya * (xc - xb)));
	   		float y = numerator/denominator;
	   		float x = (rb * rb + xa * xa + ya * ya - ra * ra - xb * xb - yb * yb - 2*(ya - yb) * y) / (2*(xa -xb));
			point ret;
			ret.x = x;
			ret.y = y;
			pts[i + j *U*V] = ret;
	   		syncthreads();
	}
}
