#include <iostream>
#include <hip/hip_runtime.h>
#include <algorithm>
#include <ctime>
#include <cmath>
#include <math.h>

#define NUM (1<<12)

using namespace std;

typedef struct{
        float x,y;
} point;

float ** generate_set(point a, point b, point c, point *points);
float get_distance(point a, point b);

int main(int argc, char *argv[]){
    srand(time(NULL));
    cout << NUM << endl;

    point *points =(point *) malloc((NUM/4) * (sizeof(point)));

    point a = {3.4,-2.4};
    point b = {5.6,1.23};
    point c = {-3.8,5.4};
    
    float ** distance_vector = generate_set(a,b,c,points);

    return 0;    
}

float ** generate_set(point a, point b, point c, point *points){

    
    float ** dist = (float **) malloc(NUM * sizeof(float *)); 
    int i,j;
    for(j = 0; j < NUM; j++){
    	  dist[j] = (float *) malloc(3 * sizeof(float));
	  for(i = 0; i < 3; i++){
	  	dist[j][i] = 0;
	}	     
    }

    srand(time(NULL));
    float x_ave = 0, y_ave = 0;
    point next;
    next.x = 0;
    next.y = 0;
    for(i = 0; i < NUM; i++){
    	  dist[i][0] = get_distance(a,next);
	  dist[i][1] = get_distance(b,next);
	  dist[i][2] = get_distance(c,next);


	  cout << dist[i][0] << "," << dist[i][1] << "," << dist[i][2] << endl;
	  
	  if(i%4 == 0 && i != 0){
	  	point t;
		t.x = x_ave/4;
		t.y = y_ave/4;
		points[(i/4)-1] = t;
		x_ave = 0;
		y_ave = 0;
	  }	  

	  x_ave	+= next.x;
	  y_ave	+= next.y;

	  //get new point
	  float temp = (rand() % 20000);
	  float delta_x = .1 - (temp / 100000);
	  temp = (rand() % 20000);
	  float delta_y = .1 - (temp / 100000);
	  
	  next.x += delta_x;
	  next.y += delta_y;
	  
    }

    return dist; 
    	  
}

float get_distance(point a, point b){
      float distance = sqrt((pow((a.x - b.x),2) + pow((a.y - b.y),2)));
      return distance;
}     