#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>

#include <algorithm>
#include <ctime>
#include <cmath>
#include <math.h>

#define NUM (1<<12)

using namespace std;

typedef struct{
        float x,y;
} point;

float ** generate_set(point a, point b, point c, point *points);
float get_distance(point a, point b);
__global__ void trilateration(point *a, point *b, point *c, float ** dv, point * pts);



int main(int argc, char *argv[]){
    srand(time(NULL));
    cout << NUM << endl;
    point *results =(point *) malloc((NUM/4) * (sizeof(point)));
    point *points =(point *) malloc((NUM/4) * (sizeof(point)));

    point a = {3.4,-2.4};
    point b = {5.6,1.23};
    point c = {-3.8,5.4};
    
    float ** distance_vector = generate_set(a,b,c,points);

    float ** dv;
    point * da;
    point * db;
    point * dc;
    point * pts;

    /*cudaMalloc(&da, sizeof(point *));
    cudaMalloc(&db, sizeof(point *));
    cudaMalloc(&dc, sizeof(point *));
    cudaMalloc((void **)&pts, (NUM/4) * sizeof(point));
    cudaMalloc((void **)&dv, NUM*sizeof(float *));

    for(int i = 0; i < NUM; i++){
    	    cudaMalloc(&dv[i], 3*sizeof(float));
    }

    cudaMemcpy(dv, distance_vector, NUM * sizeof(float*), cudaMemcpyHostToDevice);
    cudaMemcpy(da, &a, sizeof(point),cudaMemcpyHostToDevice);
    cudaMemcpy(db, &b, sizeof(point),cudaMemcpyHostToDevice);
    cudaMemcpy(dc, &c, sizeof(point),cudaMemcpyHostToDevice);*/

    hipMallocManaged(&da, sizeof(point *));
    hipMallocManaged(&db, sizeof(point *));
    hipMallocManaged(&dc, sizeof(point *));
    hipMallocManaged(&pts, (NUM/4) * sizeof(point));
    hipMallocManaged(&dv, NUM * sizeof(float *));
    /*for(int i = 0; i < NUM; i++){
	cudaMallocManaged(&dv[i], 3*sizeof(float));
    }

    *da = a;
    *db = b;
    *dc = c;
    for(int i = 0; i < NUM; i++){
	for(int j = 0; j < 3; j++){
		dv[i][j] = distance_vector[i][j];
	}
    }*/

    trilateration<<<1,1>>>(da,db,dc,dv,pts);
    hipDeviceSynchronize();
    
    //cudaMemcpy(results, pts, (NUM/4) * sizeof(point),cudaMemcpyDeviceToHost);

    /*for(int i = 0; i < NUM/4; i++){
	if(results[i].x != 0)
		cout << results[i].x << ", " << results[i].y << "\n";
    }*/

    for(int i = 0; i < NUM/4; i++){
	if(pts[i].x == 32)
		cout << pts[i].x << ", " << pts[i].y << "\n";
    }
    return 0;    
}

float ** generate_set(point a, point b, point c, point *points){

    float ** dist = (float **) malloc(NUM * sizeof(float *)); 
    int i,j;
    for(j = 0; j < NUM; j++){
    	  dist[j] = (float *) malloc(3 * sizeof(float));
	  for(i = 0; i < 3; i++){
	  	dist[j][i] = 0;
	}	     
    }

    srand(time(NULL));
    float x_ave = 0, y_ave = 0;
    point next;
    next.x = 0;
    next.y = 0;
    for(i = 0; i < NUM; i++){
    	  dist[i][0] = get_distance(a,next);
	  dist[i][1] = get_distance(b,next);
	  dist[i][2] = get_distance(c,next);


	  cout << dist[i][0] << "," << dist[i][1] << "," << dist[i][2] << endl;
	  
	  if(i%4 == 0 && i != 0){
	  	point t;
		t.x = x_ave/4;
		t.y = y_ave/4;
		points[(i/4)-1] = t;
		x_ave = 0;
		y_ave = 0;
	  }	  

	  x_ave	+= next.x;
	  y_ave	+= next.y;

	  //get new point
	  float temp = (rand() % 20000);
	  float delta_x = (temp / 100000);
	  temp = (rand() % 20000);
	  float delta_y = (temp / 100000) - .2;
	  
	  next.x += delta_x;
	  next.y += delta_y;
	  
    }

    return dist; 
    	  
}

float get_distance(point a, point b){
      float distance = sqrt((pow((a.x - b.x),2) + pow((a.y - b.y),2)));
      return distance;
}

float norm(point p){
	return pow(pow(p.x,2) + pow(p.y,2), .5);
}

__global__ void trilateration(point *a, point *b, point *c, float ** dv, point * pts){

	   int i = threadIdx.x;
	   pts[i].x = 32;
	   pts[i].y = 123;
	   /*float xa = a->x;
	   float ya = a->y;
	   float xb = b->x;
	   float yb = b->y;
	   float xc = c->x;
	   float yc = c->y;
	   float ra = dv[i][0];
	   float rb = dv[i][1];
	   float rc = dv[i][2];

	   	float S = (pow(xc, 2) - pow(xb, 2) + pow(yc, 2) - pow(yb, 2) + pow(rb, 2) - pow(rc, 2)) / 2;
		float T = (pow(xa, 2) - pow(xb, 2) + pow(ya, 2) - pow(yb, 2) + pow(ra, 2) - pow(rc, 2)) / 2;
		float y = ((T * (xb - xc)) - (S * (xb - xa))) / (((ya, yb) * (xb - xc)) - ((yc - yb) * (xb - xa)));
		float x = ((y * (ya)) - T) / (xb - xa);
		point ret;
		ret.x = x;
		ret.y = y;
		pts[i] = ret;*/

}
